#include "hip/hip_runtime.h"
#include <iostream>
#include <limits>
#include <fstream>
#include <ctime>
#include <thread>
#include <future>
#include "geometry/hitable.h"
#include "geometry/sphere.h"
#include "geometry/moving_sphere.h"
#include "geometry/hitablelist.h"
#include "camera.h"
#include "material/material.h"
#include "material/texture.h"
#include "utils/s_random.h"
#include "geometry/rectangle.h"
#include "geometry/flip.h"
#include "geometry/cornell_box.h"
#include "geometry/rotate_y.h"
#include "geometry/translate.h"
#include "geometry/triangle.h"

#include <hip/hip_runtime.h>
#define checkCudaErrors(func)				\
{									\
    hipError_t e = (func);			\
    if(e != hipSuccess)						                \
        printf ("%s %d CUDA: %s\n", __FILE__,  __LINE__, hipGetErrorString(e));		\
}

hitable *cornel_box_(){
    hitable **list = new hitable*[8];
    int i = 0;
    material *red = new lambertian(new const_texture(vec3(0.65f, 0.05f, 0.05f)));
    material *white = new lambertian(new const_texture(vec3(0.73f, 0.73f, 0.73f)));
    material *green = new lambertian(new const_texture(vec3(0.12f, 0.45f, 0.15f)));
    material *light = new diffuse_light(new const_texture(vec3(15, 15, 15)));
    list[i++] = new flip(new yz_rect(0, 555, 0, 555, 555 ,green));
    list[i++] = new yz_rect(0, 555, 0, 555, 0, red);
    list[i++] = new xz_rect(213, 343, 227, 332, 554, light);
    list[i++] = new flip(new xz_rect(0, 555, 0, 555, 555, white));
    list[i++] = new xz_rect(0, 555, 0, 555, 0, white);
    list[i++] = new flip(new xy_rect(0, 555, 0, 555, 555, white));
    list[i++] = new translate(new rotate_y(new cornell_box(vec3(0, 0 , 0), vec3(165, 165, 165), white), -18), vec3(130, 0, 65));
    list[i++] = new translate(new rotate_y(new cornell_box(vec3(0, 0 , 0), vec3(165, 330, 165), white), 15), vec3(265, 0, 295));
    return new hitable_list(list, i);
}

__host__ __device__ 
vec3 color(const ray& r, hitable *world, int depth) {
    hit_record rec;
    if (world->hit(r, 0.001f, std::numeric_limits<float>::max(), rec)) {
        ray scattered;
        vec3 attenuation;
        vec3 emitted = rec.mat_ptr->emitted(rec.u, rec.v, rec.p);
        if (depth < 20 && rec.mat_ptr->scatter(r, rec, attenuation, scattered)) {
            return emitted + attenuation * color(scattered, world, depth+1);
            // 一般物体材质不发光 第一项为0 发光物体不散射 返回亮度
        }
        else {
            return emitted;
        }
    }
    else {
        return vec3(0, 0, 0); // 环境光
    }
}

__global__ void intersect(vec3* pixel, camera* cam, hitable* world, int ny, int nx, int ns)
{
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int bszx = blockDim.x;
    int bszy = blockDim.y;
    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int i = bszy * by + ty;
    int j = bszx * bx + tx;

    vec3 col(0, 0, 0);
    for (int s = 0; s < ns; s++) {
        float u = (i + drand48()) / float(nx);
        float v = (j + drand48()) / float(ny);
        ray r = cam->get_ray(u, v);
        col += color(r, world,0);
    }
    col /= float(ns);
    col = vec3( sqrt(col[0]), sqrt(col[1]), sqrt(col[2]) );
    pixel[j * nx + i] = vec3{
        fminf(255.99f*col[0],255.0f), 
        fminf(255.99f*col[1],255.0f), 
        fminf(255.99f*col[2],255.0f)
    };
}

int main(int argc, char** argv) {
    if(argc != 2) std::cout << "please specify output filename" << std::endl;
    std::ofstream outfile;
    outfile.open(argv[1], std::ios::out);
    const int nx = 256;
    const int ny = 256;
    int ns = 20;
    std::vector<std::vector<std::vector<float>>> res(ny);
    for(int i = 0 ; i < ny ; i ++){
        res[i] = std::vector<std::vector<float>>(nx);
        for(int j = 0 ; j < nx ; j ++){
            res[i][j] = std::vector<float>(3);
        }
    }
    outfile << "P3\n" << nx << " " << ny << "\n255\n";

    hitable* world = cornel_box_();
    vec3 lookfrom = vec3(278, 278, -800);
    vec3 lookat = vec3(278, 278, 0);

    float dist_to_focus = (lookfrom - lookat).length();
    float aperture = 0;
    camera* cam = new camera(lookfrom, lookat, vec3(0,1,0), 60, float(nx) / float(ny), aperture, dist_to_focus, 0.0, 1.0);

    vec3* pixel = new vec3[ny * nx];
    
    hipEvent_t start, stop;
    float msecTotal = 0;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    dim3 dimBlock(32, ny / 32);
    dim3 dimGrid(nx / 32, ny / 32);
    intersect<<< dimGrid, dimBlock >>>(pixel, cam, world, ny, nx, ns);

    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));
    std::cout << "Total Time : " << msecTotal / 1000 / CLOCKS_PER_SEC << "s" << std::endl;

    for (int j = ny - 1 ; j >= 0 ; j --)
        for ( int i = 0 ; i < nx ; i ++ )
            outfile << static_cast<int>(pixel[j * nx + i].x()) 
            << " " << static_cast<int>(pixel[j * nx + i].y()) 
            << " " << static_cast<int>(pixel[j * nx + i].z()) << "\n";

    outfile.close();
}
